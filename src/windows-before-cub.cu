// Including windows.h before hipcub/hipcub.hpp for CUB 1.5.0 results in a compile error.
#if defined(_WIN32)
    #include <Windows.h>
    #pragma push_macro("small")
    #undef small
#endif
#include <hipcub/hipcub.hpp>
#if defined(_WIN32)
    #pragma pop_macro("small")
#endif
int main(int argc, char* argv[]) {
    return EXIT_SUCCESS;
}
